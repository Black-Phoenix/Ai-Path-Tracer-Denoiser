
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathMagma.cu"
#else

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)

#ifdef USE_MAGMA

static void THCTensor_(copyArray1d)(THCState *state, THCTensor *self, scalar_t *src, int k)
{
  int64_t size[1] = { k };
  int64_t stride[1] = { 1 };
  THCTensor_(resizeNd)(state, self, 1, size, stride);
  size_t len = k * sizeof(scalar_t);
  THCudaCheck(cudaMemcpy(THCStorage_(data)(state, THTensor_getStoragePtr(self)) + self->storage_offset(), src, len, cudaMemcpyHostToDevice));
}

static void THCTensor_(copyArray2d)(THCState *state, THCTensor *self, scalar_t *src, int m, int n)
{
  int64_t size[2] = { m, n };
  int64_t stride[2] = { 1, m };
  THCTensor_(resizeNd)(state, self, 2, size, stride);
  size_t len = m * n * sizeof(scalar_t);
  THCudaCheck(cudaMemcpy(THCStorage_(data)(state, THTensor_getStoragePtr(self)) + self->storage_offset(), src, len, cudaMemcpyHostToDevice));
}

static void THCTensor_(copyTensor2d)(THCState *state, scalar_t *dst, THCTensor *self)
{
  THAssert(self->dim() == 2);
  size_t len = THCTensor_(nElement)(state, self)*sizeof(scalar_t);
  THCTensor *temp = THCTensor_(newTranspose)(state, self, 0, 1);
  THCTensor *selfc = THCTensor_(newContiguous)(state, temp);
  THCudaCheck(cudaMemcpy(dst, THCStorage_(data)(state, THTensor_getStoragePtr(selfc)) + selfc->storage_offset(), len, cudaMemcpyDeviceToHost));
  THCTensor_(free)(state, temp);
  THCTensor_(free)(state, selfc);
}

#endif // USE_MAGMA

static THCTensor* THCTensor_(newColumnMajor)(THCState *state, THCTensor *self, THCTensor *src)
{
  THAssert(src->dim() == 2);
  if (self == src && self->stride(0) == 1 && self->stride(1) == self->size(0))
  {
    THCTensor_(retain)(state, self);
    return self;
  }

  if (self == src)
    self = THCTensor_(new)(state);
  else
    THCTensor_(retain)(state, self);

  int64_t size[2] = { src->size(0), src->size(1) };
  int64_t stride[2] = { 1, src->size(0) };

  THCTensor_(resizeNd)(state, self, 2, size, stride);
  THCTensor_(copy)(state, self, src);
  return self;
}

void THCTensor_(gels)(THCState *state, THCTensor *rb_, THCTensor *ra_, THCTensor *b_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 1, "A should be (non-empty) 2 dimensional");
  THArgCheck(!b_->is_empty() && b_->dim() == 2, 1, "b should be (non-empty) 2 dimensional");
  TORCH_CHECK(a_->size(0) == b_->size(0), "Expected A and b to have same size "
      "at dim 0, but A has ", a_->size(0), " rows and B has ", b_->size(0), " rows");
  THArgCheck(a_->size(0) >= a_->size(1), 2, "Expected A with shape (m x n) to have "
      "m >= n. The case for m < n is not implemented yet.");

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  THCTensor *b = THCTensor_(newColumnMajor)(state, rb_, b_);
  scalar_t *a_data = THCTensor_(data)(state, a);
  scalar_t *b_data = THCTensor_(data)(state, b);

  int64_t m = a->size(0);
  int64_t n = a->size(1);
  int64_t nrhs = b->size(1);
  scalar_t wkopt;

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, &wkopt, -1, &info);
#else
  magma_dgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, &wkopt, -1, &info);
#endif

  scalar_t *hwork = th_magma_malloc_pinned<scalar_t>((size_t)wkopt);

#if defined(THC_REAL_IS_FLOAT)
  magma_sgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, hwork, (int)wkopt, &info);
#else
  magma_dgels_gpu(MagmaNoTrans, m, n, nrhs, a_data, m, b_data, m, hwork, (int)wkopt, &info);
#endif

  magma_free_pinned(hwork);

  if (info != 0)
    THError("MAGMA gels : Argument %d : illegal value", -info);

  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(freeCopyTo)(state, b, rb_);
#else
  THError(NoMagma(gels));
#endif
}

void THCTensor_(geev)(THCState *state, THCTensor *re_, THCTensor *rv_, THCTensor *a_, bool eigenvectors)
{
#ifdef USE_MAGMA
  char jobvrs = eigenvectors ? 'V' : 'N';
  THArgCheck(a_->dim() == 2, 3, "A should be 2 dimensional");
  THArgCheck(a_->size(0) == a_->size(1), 3, "A should be square");

  magma_vec_t jobvr = jobvrs == 'N' ? MagmaNoVec : MagmaVec;
  int64_t n = a_->size(0);

  scalar_t *a_data = th_magma_malloc_pinned<scalar_t>(n * n);
  THCTensor_(copyTensor2d)(state, a_data, a_);

  scalar_t *wr = th_magma_malloc_pinned<scalar_t>(n);
  scalar_t *wi = th_magma_malloc_pinned<scalar_t>(n);

  scalar_t *vr_data = NULL;
  int64_t ldvr = 1;
  if (jobvr == MagmaVec)
  {
    vr_data = th_magma_malloc_pinned<scalar_t>(n * n);
    ldvr = n;
  }

  scalar_t *work_data = nullptr;

  if (n > 0) {
    int info;
    scalar_t wkopt;
#if defined(THC_REAL_IS_FLOAT)
    magma_sgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, &wkopt, -1, &info);
#else
    magma_dgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, &wkopt, -1, &info);
#endif

    int lwork = (int) wkopt;
    work_data = th_magma_malloc_pinned<scalar_t>(lwork);

#if defined(THC_REAL_IS_FLOAT)
    magma_sgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, work_data, lwork, &info);
#else
    magma_dgeev(MagmaNoVec, jobvr, n, a_data, n, wr, wi, NULL, 1, vr_data, ldvr, work_data, lwork, &info);
#endif

    if (info > 0)
      THError("MAGMA geev : Failed to converge. %d off-diagonal elements of an didn't converge to zero", info);
    else if (info < 0)
      THError("MAGMA geev : Argument %d : illegal value", -info);
  }

  {
    THCTensor_(resize2d)(state, re_, 2, n);
    THCTensor *re = THCTensor_(newContiguous)(state, re_);
    if (n > 0) {
      THCudaCheck(cudaMemcpy(THCStorage_(data)(state, THTensor_getStoragePtr(re)) + re->storage_offset(), wr, n*sizeof(scalar_t), cudaMemcpyHostToDevice));
      THCudaCheck(cudaMemcpy(THCStorage_(data)(state, THTensor_getStoragePtr(re)) + re->storage_offset() + n, wi, n*sizeof(scalar_t), cudaMemcpyHostToDevice));
    }
    THCTensor_(freeCopyTo)(state, re, re_);
    THCTensor_(transpose)(state, re_, NULL, 0, 1);
  }

  if (jobvr == MagmaVec)
    THCTensor_(copyArray2d)(state, rv_, vr_data, n, n);

  magma_free_pinned(work_data);
  magma_free_pinned(vr_data);
  magma_free_pinned(wi);
  magma_free_pinned(wr);
  magma_free_pinned(a_data);

#else
  THError(NoMagma(geev));
#endif
}

__global__ void THCTensor_(copyUpperSymmetric)(scalar_t *input, int n, int len)
{
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += 65535) {
    const int r = idx % n;
    const int c = idx / n;
    if (r > c) {
      input[idx] = input[r*n + c];
    }
  }
}

__global__ void THCTensor_(copyLowerSymmetric)(scalar_t *input, int n, int len)
{
  for (int idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += 65535) {
    const int r = idx % n;
    const int c = idx / n;
    if (r < c) {
      input[idx] = input[r*n + c];
    }
  }
}

void THCTensor_(potri)(THCState *state, THCTensor *ra_, THCTensor *a, bool upper)
{
  char uplo = upper ? 'U' : 'L';
#ifdef USE_MAGMA
  THArgCheck(!a->is_empty() && a->dim() == 2, 2, "A should be non-empty 2 dimensional");
  THArgCheck(a->size(0) == a->size(1), 2, "A should be square");

  int64_t n = a->size(0);
  magma_uplo_t ul = uplo == 'U' ?  MagmaUpper : MagmaLower;

  THCTensor *input = THCTensor_(newColumnMajor)(state, ra_, a);
  scalar_t *input_data = THCTensor_(data)(state, input);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_spotri_gpu(ul, n, input_data, n, &info);
#else
  magma_dpotri_gpu(ul, n, input_data, n, &info);
#endif

  if (info > 0)
    THError("MAGMA potri : A(%d,%d) is 0, A cannot be factorized", info, info);
  else if (info < 0)
    THError("MAGMA potri : Argument %d : illegal value", -info);

  cudaStream_t stream = THCState_getCurrentStream(state);
  const int len = n*n;
  dim3 blocks(std::min(DIVUP(len, 128), 65535));
  dim3 threads(128);
  if (uplo == 'U') {
    THCTensor_(copyUpperSymmetric)<<<blocks, threads, 0, stream>>>(input_data, n, len);
  } else {
    THCTensor_(copyLowerSymmetric)<<<blocks, threads, 0, stream>>>(input_data, n, len);
  }

  THCTensor_(freeCopyTo)(state, input, ra_);
#else
  THError(NoMagma(potri));
#endif
}

void THCTensor_(geqrf)(THCState *state, THCTensor *ra_, THCTensor *rtau_, THCTensor *a_)
{
#ifdef USE_MAGMA
  THArgCheck(!a_->is_empty() && a_->dim() == 2, 2, "A should be non-empty 2 dimensional");

  THCTensor *a = THCTensor_(newColumnMajor)(state, ra_, a_);
  int64_t m = a->size(0);
  int64_t n = a->size(1);
  int64_t k = (m < n ? m : n);

#if defined(THC_REAL_IS_FLOAT)
  int64_t nb = magma_get_sgeqrf_nb(m, n);
#else
  int64_t nb = magma_get_dgeqrf_nb(m, n);
#endif

  scalar_t *rtau_data = th_magma_malloc_pinned<scalar_t>(k);
  scalar_t *a_data = THCTensor_(data)(state, a);

  int info;
#if defined(THC_REAL_IS_FLOAT)
  magma_sgeqrf2_gpu(m, n, a_data, m, rtau_data, &info);
#else
  magma_dgeqrf2_gpu(m, n, a_data, m, rtau_data, &info);
#endif

  if (info != 0)
    THError("MAGMA geqrf2 : Argument %d : illegal value.", -info);

  THCTensor_(freeCopyTo)(state, a, ra_);
  THCTensor_(copyArray1d)(state, rtau_, rtau_data, k);
  magma_free_pinned(rtau_data);
#else
  THError(NoMagma(geqrf));
#endif
}

#endif

#endif
