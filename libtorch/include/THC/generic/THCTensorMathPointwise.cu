
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorMathPointwise.cu"
#else

#include <ATen/MemoryOverlap.h>
#include <ATen/NamedTensorUtils.h>
#include <ATen/core/EnableNamedTensor.h>

void THCTensor_(cbitand)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  return THError("cbitand is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorBitAndOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorBitAndOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(cbitor)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  return THError("cbitor is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorBitOrOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorBitOrOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(cbitxor)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF) || defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE)
  return THError("cbitor is only supported for integer type tensors");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorBitXorOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorBitXorOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(cmax)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorMaxOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorMaxOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cmin)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorMinOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorMinOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cmaxValue)(THCState *state, THCTensor *self, THCTensor *src, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self, TensorMaxValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src, TensorMaxValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cminValue)(THCState *state, THCTensor *self, THCTensor *src, scalar_t value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self, src));

  if (self == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self, TensorMinValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src);
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src, TensorMinValueOp<scalar_t>(value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

#if !defined(THC_REAL_IS_BOOL)

static void propagate_names_if_named_tensor_enabled(THCTensor* result, THCTensor* src) {
#ifdef BUILD_NAMEDTENSOR
  at::namedinference::propagate_names(result, src);
#endif
}

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)             \
  struct Tensor_##NAME##_##REAL##_Op {                                  \
    __device__ __forceinline__ void operator()(scalar_t* out, scalar_t* in) const { \
      *out = CFUNC(*in);                                                \
    }                                                                   \
                                                                        \
    __device__ __forceinline__ void operator()(scalar_t* v) const {         \
      *v = CFUNC(*v);                                                   \
    }                                                                   \
  };                                                                    \
                                                                        \
  void THCTensor_(NAME)(THCState* state, THCTensor* self_, THCTensor* src) { \
    THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));       \
    at::assert_no_internal_overlap(self_);                              \
    if (self_ == src) {                                                 \
      if (!THC_pointwiseApply1<scalar_t>(state, self_, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    } else {                                                            \
      THCTensor_(resizeAs)(state, self_, src);                          \
                                                                        \
      if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, Tensor_##NAME##_##REAL##_Op())) { \
        THArgCheck(false, 2, CUTORCH_DIM_WARNING);                      \
      }                                                                 \
    }                                                                   \
                                                                        \
    THCudaCheck(cudaGetLastError());                                    \
    propagate_names_if_named_tensor_enabled(self_, src);                \
  }

#define IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(NAME, CFUNC, REAL) \
  IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_(NAME, CFUNC, REAL)

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log10, THCNumerics<scalar_t>::log10, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(log1p, THCNumerics<scalar_t>::log1p, Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( log2, THCNumerics<scalar_t>::log2,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  exp, THCNumerics<scalar_t>::exp,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cos, THCNumerics<scalar_t>::cos,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  sin, THCNumerics<scalar_t>::sin,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC( sqrt, THCNumerics<scalar_t>::sqrt,  Real)

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  acos, THCNumerics<scalar_t>::acos,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cosh, THCNumerics<scalar_t>::cosh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  asin, THCNumerics<scalar_t>::asin,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  sinh, THCNumerics<scalar_t>::sinh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(   tan, THCNumerics<scalar_t>::tan,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  atan, THCNumerics<scalar_t>::atan,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  tanh, THCNumerics<scalar_t>::tanh,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(   erf, THCNumerics<scalar_t>::erf,   Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  erfc, THCNumerics<scalar_t>::erfc,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  frac, THCNumerics<scalar_t>::frac,  Real)
IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  cinv, THCNumerics<scalar_t>::cinv,  Real)

#endif

IMPLEMENT_CUDA_TENSOR_BASIC_FUNC(  abs, THCNumerics<scalar_t>::abs,   Real)

#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC_
#undef IMPLEMENT_CUDA_TENSOR_BASIC_FUNC

void THCTensor_(clamp)(THCState *state, THCTensor *self_, THCTensor *src, scalar_t min_value,
  scalar_t max_value)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorClampOp<scalar_t>(min_value, max_value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorClampOp<scalar_t>(min_value, max_value))) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
}

void THCTensor_(crossKernel)(THCState *state, THCTensor *self, THCTensor *x, THCTensor *y, int dimension)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, x, y));

  int64_t sx = THCTensor_(stride)(state, x, dimension);
  int64_t sy = THCTensor_(stride)(state, y, dimension);
  int64_t so = THCTensor_(stride)(state, self, dimension);
  THCTensor *nx = THCTensor_(newNarrow)(state, x, dimension, 0, 1);
  THCTensor *ny = THCTensor_(newNarrow)(state, y, dimension, 0, 1);
  THCTensor *nself = THCTensor_(newNarrow)(state, self, dimension, 0, 1);
  if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, nself, nx, ny, TensorCrossOp<scalar_t>(sx, sy, so))) {
    THArgCheck(false, 2, CUTORCH_DIM_WARNING);
  }
  THCTensor_(free)(state, nx);
  THCTensor_(free)(state, ny);
  THCTensor_(free)(state, nself);
}

#if defined(THC_REAL_IS_FLOAT) || defined(THC_REAL_IS_DOUBLE) || defined(THC_REAL_IS_HALF)

void THCTensor_(sigmoid)(THCState* state, THCTensor* self_, THCTensor* src) {
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 2, self_, src));
  if (self_ == src) {
    if (!THC_pointwiseApply1<scalar_t>(state, self_, TensorSigmoidOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src);

    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src, TensorSigmoidOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#ifdef BUILD_NAMEDTENSOR
  at::namedinference::propagate_names(self_, src);
#endif
}

#endif

namespace {
c10::intrusive_ptr<at::TensorImpl, at::UndefinedTensorImpl> retainTensorImpl(THCTensor* self) {
  c10::raw::intrusive_ptr::incref(self);
  return c10::intrusive_ptr<at::TensorImpl, at::UndefinedTensorImpl>::reclaim(self);
}
}

void THCTensor_(cadd)(THCState *state, THCTensor *self_, THCTensor* src1, scalar_t value, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
#ifdef THC_REAL_IS_HALF
  auto alpha = at::Half(value);
#else
  auto alpha = value;
#endif
  at::add_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)), alpha);
}

void THCTensor_(csub)(THCState *state, THCTensor *self_, THCTensor* src1, scalar_t value, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
#ifdef THC_REAL_IS_HALF
  auto alpha = at::Half(value);
#else
  auto alpha = value;
#endif
  at::sub_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)), alpha);
}

void THCTensor_(cmul)(THCState *state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
  at::mul_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)));
}

void THCTensor_(cdiv)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
  auto out = at::Tensor(retainTensorImpl(self_));
  at::div_out(out, at::Tensor(retainTensorImpl(src1)), at::Tensor(retainTensorImpl(src2)));
}

void THCTensor_(clshift)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF)
  return THError("clshift not supported for torch.CudaHalfTensor");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorLShiftOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorLShiftOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(crshift)(THCState* state, THCTensor *self_, THCTensor *src1, THCTensor *src2)
{
#if defined(THC_REAL_IS_HALF)
  return THError("crshift not supported for torch.CudaHalfTensor");
#else
  THAssert(THCTensor_(checkGPU)(state, 3, self_, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 3, "sizes do not match");

  if (self_ == src1) {
    // self /= src2
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self_, src2, TensorRShiftOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self_, src1);

    // self = src1 / src2
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self_, src1, src2, TensorRShiftOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }

  THCudaCheck(cudaGetLastError());
#endif
}

void THCTensor_(cremainder)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorCRemainderOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorCRemainderOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

void THCTensor_(cfmod)(THCState *state, THCTensor *self, THCTensor *src1, THCTensor *src2)
{
  THCAssertSameGPU(THCTensor_(checkGPU)(state, 3, self, src1, src2));
  THArgCheck(THCTensor_(nElement)(state, src1) ==
             THCTensor_(nElement)(state, src2), 2, "sizes do not match");

  if (self == src1) {
    if (!THC_pointwiseApply2<scalar_t, scalar_t>(state, self, src2, TensorCFmodOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  } else {
    THCTensor_(resizeAs)(state, self, src1);
    if (!THC_pointwiseApply3<scalar_t, scalar_t, scalar_t>(state, self, src1, src2, TensorCFmodOp<scalar_t>())) {
      THArgCheck(false, 2, CUTORCH_DIM_WARNING);
    }
  }
}

#endif
#endif
